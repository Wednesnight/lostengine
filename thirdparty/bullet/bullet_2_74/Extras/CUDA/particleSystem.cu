#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2006 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and 
 * international Copyright laws.  
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE 
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH 
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF 
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.   
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL, 
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS 
 * OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE 
 * OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE 
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.  This source code is a "commercial item" as 
 * that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of 
 * "commercial computer software" and "commercial computer software 
 * documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995) 
 * and is provided to the U.S. Government only as a commercial end item.  
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through 
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

//#include <cutil.h>
#include <cstdlib>
#include <cstdio>
#include <string.h>

#if defined(__APPLE__) || defined(MACOSX)
#include <GLUT/glut.h>
#else
#include <GL/glut.h>
#endif

#include <cuda_gl_interop.h>

#include "particles_kernel.cu"
//#include "radixsort.cu"

    //! Check for CUDA error
#  define CUT_CHECK_ERROR(errorMessage) do {                                 \
    hipError_t err = hipGetLastError();                                    \
    if( hipSuccess != err) {                                                \
        fprintf(stderr, "Cuda error: %s in file '%s' in line %i : %s.\n",    \
                errorMessage, __FILE__, __LINE__, hipGetErrorString( err) );\
        mm_exit(EXIT_FAILURE);                                                  \
    }                                                                        \
    err = hipDeviceSynchronize();                                           \
    if( hipSuccess != err) {                                                \
        fprintf(stderr, "Cuda error: %s in file '%s' in line %i : %s.\n",    \
                errorMessage, __FILE__, __LINE__, hipGetErrorString( err) );\
        mm_exit(EXIT_FAILURE);                                                  \
    } } while (0)


#  define MY_CUDA_SAFE_CALL_NO_SYNC( call) do {                                 \
    hipError_t err = call;                                                    \
    if( hipSuccess != err) {                                                \
        fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",        \
                __FILE__, __LINE__, hipGetErrorString( err) );              \
        mm_exit(EXIT_FAILURE);                                                  \
    } } while (0)

#  define MY_CUDA_SAFE_CALL( call) do {                                         \
    MY_CUDA_SAFE_CALL_NO_SYNC(call);                                            \
    hipError_t err = hipDeviceSynchronize();                                 \
    if( hipSuccess != err) {                                                \
        fprintf(stderr, "Cuda errorSync in file '%s' in line %i : %s.\n",        \
                __FILE__, __LINE__, hipGetErrorString( err) );              \
        mm_exit(EXIT_FAILURE);                                                  \
    } } while (0)



extern "C"
{
void mm_exit(int val)
{
	exit(val);
}

void cudaInit(int argc, char **argv)
{   
//    CUT_DEVICE_INIT(argc, argv);
}

void allocateArray(void **devPtr, size_t size)
{
    MY_CUDA_SAFE_CALL(hipMalloc(devPtr, size));
}

void freeArray(void *devPtr)
{
    MY_CUDA_SAFE_CALL(hipFree(devPtr));
}

void threadSync()
{
    MY_CUDA_SAFE_CALL(hipDeviceSynchronize());
}

void copyArrayFromDevice(void* host, const void* device, unsigned int vbo, int size)
{   
    if (vbo)
        MY_CUDA_SAFE_CALL(cudaGLMapBufferObject((void**)&device, vbo));
    MY_CUDA_SAFE_CALL(hipMemcpy(host, device, size, hipMemcpyDeviceToHost));
    if (vbo)
        MY_CUDA_SAFE_CALL(cudaGLUnmapBufferObject(vbo));
}

void copyArrayToDevice(void* device, const void* host, int offset, int size)
{
    MY_CUDA_SAFE_CALL(hipMemcpy((char *) device + offset, host, size, hipMemcpyHostToDevice));
}

void registerGLBufferObject(uint vbo)
{
    MY_CUDA_SAFE_CALL(cudaGLRegisterBufferObject(vbo));
}

void unregisterGLBufferObject(uint vbo)
{
    MY_CUDA_SAFE_CALL(cudaGLUnregisterBufferObject(vbo));
}

void setParameters(SimParams *hostParams)
{
    // copy parameters to constant memory
    MY_CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(params), hostParams, sizeof(SimParams)) );
}

//Round a / b to nearest higher integer value
int iDivUp(int a, int b){
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

// compute grid and thread block size for a given number of elements
void computeGridSize(int n, int blockSize, int &numBlocks, int &numThreads)
{
    numThreads = min(blockSize, n);
    numBlocks = iDivUp(n, numThreads);
}

void 
integrateSystem(uint vboOldPos, uint vboNewPos, 
                float* oldVel, float* newVel, 
                float deltaTime,
                int numBodies)
{
    int numThreads, numBlocks;
    computeGridSize(numBodies, 256, numBlocks, numThreads);

    float *oldPos, *newPos;
    MY_CUDA_SAFE_CALL(cudaGLMapBufferObject((void**)&oldPos, vboOldPos));
    MY_CUDA_SAFE_CALL(cudaGLMapBufferObject((void**)&newPos, vboNewPos));

    // execute the kernel
    integrate<<< numBlocks, numThreads >>>((float4*)newPos, (float4*)newVel,
                                           (float4*)oldPos, (float4*)oldVel,
                                           deltaTime);
    
    // check if kernel invocation generated an error
    CUT_CHECK_ERROR("integrate kernel execution failed");

    MY_CUDA_SAFE_CALL(cudaGLUnmapBufferObject(vboOldPos));
    MY_CUDA_SAFE_CALL(cudaGLUnmapBufferObject(vboNewPos));
}

void 
updateGrid(uint    vboPos, 
           uint*   gridCounters,
           uint*   gridCells,
           uint    numBodies,
           uint    numCells)
{
    int numThreads, numBlocks;
    computeGridSize(numBodies, 256, numBlocks, numThreads);

    float *pos;
    MY_CUDA_SAFE_CALL(cudaGLMapBufferObject((void**)&pos, vboPos));

    MY_CUDA_SAFE_CALL(hipMemset(gridCounters, 0, numCells*sizeof(uint)));

    // execute the kernel
    updateGridD<<< numBlocks, numThreads >>>((float4 *) pos,
                                             gridCounters,
                                             gridCells);
    
    // check if kernel invocation generated an error
    CUT_CHECK_ERROR("Kernel execution failed");

    MY_CUDA_SAFE_CALL(cudaGLUnmapBufferObject(vboPos));
}


void 
calcHash(uint    vboPos, 
         uint*   particleHash,
         int     numBodies)
{
    int numThreads, numBlocks;
    computeGridSize(numBodies, 256, numBlocks, numThreads);

    float *pos;
    MY_CUDA_SAFE_CALL(cudaGLMapBufferObject((void**)&pos, vboPos));

    // execute the kernel
    calcHashD<<< numBlocks, numThreads >>>((float4 *) pos,
                                           (uint2 *) particleHash);
    
    // check if kernel invocation generated an error
    CUT_CHECK_ERROR("Kernel execution failed");

    MY_CUDA_SAFE_CALL(cudaGLUnmapBufferObject(vboPos));
}

void 
reorderDataAndFindCellStart(uint*  particleHash,
							uint   vboOldPos,
							float* oldVel,
							float* sortedPos,
							float* sortedVel,
							uint*  cellStart,
							uint   numBodies,
							uint   numCells)
{
    int numThreads, numBlocks;
    computeGridSize(numBodies, 256, numBlocks, numThreads);

	MY_CUDA_SAFE_CALL(hipMemset(cellStart, 0xffffffff, numCells*sizeof(uint)));

    float *oldPos;
    MY_CUDA_SAFE_CALL(cudaGLMapBufferObject((void**)&oldPos, vboOldPos));

#if USE_TEX
    MY_CUDA_SAFE_CALL(hipBindTexture(0, oldPosTex, oldPos, numBodies*sizeof(float4)));
    MY_CUDA_SAFE_CALL(hipBindTexture(0, oldVelTex, oldVel, numBodies*sizeof(float4)));
#endif

    reorderDataAndFindCellStartD<<< numBlocks, numThreads >>>(
		(uint2 *)  particleHash,
        (float4 *) oldPos,
        (float4 *) oldVel,
        (float4 *) sortedPos,
        (float4 *) sortedVel,
        (uint *)   cellStart);
    CUT_CHECK_ERROR("Kernel execution failed: reorderDataAndFindCellStartD");

#if USE_TEX
    MY_CUDA_SAFE_CALL(hipUnbindTexture(oldPosTex));
    MY_CUDA_SAFE_CALL(hipUnbindTexture(oldVelTex));
#endif

    MY_CUDA_SAFE_CALL(cudaGLUnmapBufferObject(vboOldPos));
}

void
collide(uint   vboOldPos, uint vboNewPos,
        float* sortedPos, float* sortedVel,
        float* oldVel, float* newVel,
        uint*  gridCounters,
        uint*  gridCells,
        uint*  particleHash,
        uint*  cellStart,
        uint   numBodies,
        uint   numCells,
        uint   maxParticlesPerCell)
{
    float4 *oldPos, *newPos;
    MY_CUDA_SAFE_CALL(cudaGLMapBufferObject((void**)&oldPos, vboOldPos));
    MY_CUDA_SAFE_CALL(cudaGLMapBufferObject((void**)&newPos, vboNewPos));

#if USE_TEX

#if USE_SORT
    // use sorted arrays
    MY_CUDA_SAFE_CALL(hipBindTexture(0, oldPosTex, sortedPos, numBodies*sizeof(float4)));
    MY_CUDA_SAFE_CALL(hipBindTexture(0, oldVelTex, sortedVel, numBodies*sizeof(float4)));

    MY_CUDA_SAFE_CALL(hipBindTexture(0, particleHashTex, particleHash, numBodies*sizeof(uint2)));
    MY_CUDA_SAFE_CALL(hipBindTexture(0, cellStartTex, cellStart, numCells*sizeof(uint)));
#else

    MY_CUDA_SAFE_CALL(hipBindTexture(0, oldPosTex, oldPos, numBodies*sizeof(float4)));
    MY_CUDA_SAFE_CALL(hipBindTexture(0, oldVelTex, oldVel, numBodies*sizeof(float4)));

    MY_CUDA_SAFE_CALL(hipBindTexture(0, gridCountersTex, gridCounters,numCells*sizeof(uint)));
    MY_CUDA_SAFE_CALL(hipBindTexture(0, gridCellsTex, gridCells, numCells*maxParticlesPerCell*sizeof(uint)));
#endif

#endif

    // thread per particle
    int numThreads, numBlocks;
    computeGridSize(numBodies, BLOCKDIM, numBlocks, numThreads);

    // execute the kernel
    collideD<<< numBlocks, numThreads >>>((float4*)newPos, (float4*)newVel,
#if USE_SORT
                                          (float4*)sortedPos, (float4*)sortedVel,
                                          (uint2 *) particleHash,
                                          cellStart
#else
                                          (float4*)oldPos, (float4*)oldVel,
                                          gridCounters,
                                          gridCells
#endif
                                          );

    // check if kernel invocation generated an error
    CUT_CHECK_ERROR("Kernel execution failed");

    MY_CUDA_SAFE_CALL(cudaGLUnmapBufferObject(vboNewPos));
    MY_CUDA_SAFE_CALL(cudaGLUnmapBufferObject(vboOldPos));

#if USE_TEX
    MY_CUDA_SAFE_CALL(hipUnbindTexture(oldPosTex));
    MY_CUDA_SAFE_CALL(hipUnbindTexture(oldVelTex));

#if USE_SORT
    MY_CUDA_SAFE_CALL(hipUnbindTexture(particleHashTex));
    MY_CUDA_SAFE_CALL(hipUnbindTexture(cellStartTex));
#else
    MY_CUDA_SAFE_CALL(hipUnbindTexture(gridCountersTex));
    MY_CUDA_SAFE_CALL(hipUnbindTexture(gridCellsTex));
#endif
#endif
}

}   // extern "C"
