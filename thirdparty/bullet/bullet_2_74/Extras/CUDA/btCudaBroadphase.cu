#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2006 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and 
 * international Copyright laws.  
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE 
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH 
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF 
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.   
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL, 
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS 
 * OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE 
 * OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE 
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.  This source code is a "commercial item" as 
 * that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of 
 * "commercial computer software" and "commercial computer software 
 * documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995) 
 * and is provided to the U.S. Government only as a commercial end item.  
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through 
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

#include <cstdlib>
#include <cstdio>
#include <string.h>

#include "cutil_math.h"
#include "hip/hip_math_constants.h"

#if defined(__APPLE__) || defined(MACOSX)
#include <GLUT/glut.h>
#else
#include <GL/glut.h>
#endif

#include <cuda_gl_interop.h>

#include "btCudaBroadphaseKernel.h"
//#include "radixsort.cu"


//----------------------------------------------------------------------------------------
//----------------------------------------------------------------------------------------
//----------------------------------------------------------------------------------------
//----------------------------------------------------------------------------------------
//               K E R N E L    F U N C T I O N S 
//----------------------------------------------------------------------------------------
//----------------------------------------------------------------------------------------
//----------------------------------------------------------------------------------------
//----------------------------------------------------------------------------------------

#ifdef __DEVICE_EMULATION__
	#define B_CUDA_USE_TEX 0
#else
	#define B_CUDA_USE_TEX 1
#endif

__device__ inline btCuda3F1U tex_fetch3F1U(float4 a) { return *((btCuda3F1U*)(&a)); }

#if B_CUDA_USE_TEX
	#define BT3DGRIDFETCH(t, i) tex_fetch3F1U(tex1Dfetch(t##Tex, i))
#else
	#define BT3DGRIDFETCH(t, i) t[i]
#endif

texture<uint2, 1, hipReadModeElementType> particleHashTex;
texture<uint, 1, hipReadModeElementType> cellStartTex;
texture<float4, 1, hipReadModeElementType> pAABBTex;

//----------------------------------------------------------------------------------------

__constant__ btCudaBroadphaseParams params;

//----------------------------------------------------------------------------------------

#define BT3DGRID__device__ __device__
#define BT3DGRIDmax(a, b) max(a, b)
#define BT3DGRIDmin(a, b) min(a, b)
#define BT3DGRIDparams params
#define BT3DGRID__mul24(a, b) __mul24(a, b)
#define BT3DGRID__global__ __global__
#define BT3DGRID__shared__ __shared__
#define BT3DGRID__syncthreads() __syncthreads()
#define BT3DGRIDmake_uint2(x, y) make_uint2(x, y)
#define BT3DGRIDmake_int3(x, y, z) make_int3(x, y, z)
#define BT3DGRIDPREF(func) btCuda_##func
#define BT3DGPRDMemset hipMemset
#define BT3DGRIDblockIdx blockIdx
#define BT3DGRIDblockDim blockDim
#define BT3DGRIDthreadIdx threadIdx
#define BT3DGRIDEXECKERNEL(numb, numt, kfunc, args) kfunc<<<numb, numt>>>args

//----------------------------------------------------------------------------------------

//! Check for CUDA error
#  define CUT_CHECK_ERROR(errorMessage) do {                                 \
    hipError_t err = hipGetLastError();                                    \
    if( hipSuccess != err) {                                                \
        fprintf(stderr, "Cuda error: %s in file '%s' in line %i : %s.\n",    \
                errorMessage, __FILE__, __LINE__, hipGetErrorString( err) );\
        btCuda_exit(EXIT_FAILURE);                                           \
    }                                                                        \
    err = hipDeviceSynchronize();                                           \
    if( hipSuccess != err) {                                                \
        fprintf(stderr, "Cuda error: %s in file '%s' in line %i : %s.\n",    \
                errorMessage, __FILE__, __LINE__, hipGetErrorString( err) );\
        btCuda_exit(EXIT_FAILURE);                                           \
    } } while (0)


#  define MY_CUDA_SAFE_CALL_NO_SYNC( call) do {                              \
    hipError_t err = call;                                                    \
    if( hipSuccess != err) {                                                \
        fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",        \
                __FILE__, __LINE__, hipGetErrorString( err) );              \
        btCuda_exit(EXIT_FAILURE);                                           \
    } } while (0)

#  define MY_CUDA_SAFE_CALL( call) do {                                      \
    MY_CUDA_SAFE_CALL_NO_SYNC(call);                                         \
    hipError_t err = hipDeviceSynchronize();                                 \
    if( hipSuccess != err) {                                                \
        fprintf(stderr, "Cuda errorSync in file '%s' in line %i : %s.\n",    \
                __FILE__, __LINE__, hipGetErrorString( err) );              \
        btCuda_exit(EXIT_FAILURE);                                           \
    } } while (0)

//----------------------------------------------------------------------------------------

void btCuda_exit(int val)
{
	exit(val);
}

void btCuda_allocateArray(void** devPtr, unsigned int size)
{
    MY_CUDA_SAFE_CALL(hipMalloc(devPtr, size));
}

void btCuda_freeArray(void* devPtr)
{
    MY_CUDA_SAFE_CALL(hipFree(devPtr));
}

void btCuda_copyArrayFromDevice(void* host, const void* device, unsigned int size)
{   
    MY_CUDA_SAFE_CALL(hipMemcpy(host, device, size, hipMemcpyDeviceToHost));
}

void btCuda_copyArrayToDevice(void* device, const void* host, unsigned int size)
{
    MY_CUDA_SAFE_CALL(hipMemcpy((char*)device, host, size, hipMemcpyHostToDevice));
}

void btCuda_setParameters(btCudaBroadphaseParams* hostParams)
{
    // copy parameters to constant memory
    MY_CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(params), hostParams, sizeof(btCudaBroadphaseParams)));
}

//----------------------------------------------------------------------------------------

#include "bt3DGridBroadphaseFunc.h"

//----------------------------------------------------------------------------------------

